#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

#include"include/cupdpu.cuh"
#include"include/double2posit.cuh"
#include"include/posit.cuh"
#include"include/rw_bin.cuh"
#include"include/cupdpu.cuh"
#include"SoftPositE.h"

using namespace std;

//A*B+C 功能测试
int main_ptc(){
    PositTensorCore M;
    uint32_t a[16];
    uint32_t b[16];
    uint64_t c[16];
    for(int i=0;i<16;i++){
        a[i] = 0x40000000;
        b[i] = 0x40000000;
        c[i] = 0x40000000;
    }    
    M.init(a,b,c);
    M.a_mul_b();
    matrix_print<uint32_t>(M.a);
    matrix_print<uint32_t>(M.b);
    matrix_print<uint64_t>(M.out);
    M.mul_add();
    matrix_print<uint64_t>(M.c);
    matrix_print<uint64_t>(M.out);
    return 0;
}

//mdu test
int main_mdu_test(){
    double wgts_data[147];
    double acts_data[147];
    double outs_data;
    int row = 38;
    int column = 12500;
    double mse = 0;

    uint32_t *a,*b;
    posit32_t posit;

    a = (uint32_t*)malloc(N * sizeof(uint32_t));
    b = (uint32_t*)malloc(N * sizeof(uint32_t));
    uint32_t acc[1] = {0};
    posit32_t posit_out;
    for(row=1;row<64;row++){
        for(column=1;column<4;column++){
            //读取表格中数据
            read_wgts_mat_row(row,wgts_data);
            read_acts_mat_column(column,acts_data);
            outs_data = read_outs_mat_row_column(row,column);

            //将double数据转换为posit格式并存入ab中
            for(int i=0;i<N;i++){
                posit = convertDoubleToPosit(wgts_data[i],in_BITS,ES);
                a[i] = posit.v;
                posit = convertDoubleToPosit(acts_data[i],in_BITS,ES);
                b[i] = posit.v;
            }
            acc[0] = 0;
            posit_out.v = mac_dpu(a,b,acc);
            mse += abs(convertPositToDouble(posit_out,out_BITS,ES)-outs_data);
            // cout<<"原始数据"<<outs_data<<endl;
            // cout<<"输出"<<convertPositToDouble(posit_out,out_BITS,ES)<<endl;
        }
        cout<<row<<"/63"<<endl;
    }
    mse /= (63*3);
    cout << mse << endl;
    return 0;
}

//pdpu
int main_pdpu(){
    double wgts_data[147];
    double acts_data[147];
    double outs_data;
    int row = 38;
    int column = 12500;

    for(row=0;row<64;row++){
        for(column=0;column<12544;column++){

        }
    }
    //读取表格中数据
    read_wgts_mat_row(row,wgts_data);
    read_acts_mat_column(column,acts_data);
    outs_data = read_outs_mat_row_column(row,column);

    uint32_t *a,*b;
    posit32_t posit;

    a = (uint32_t*)malloc(N * sizeof(uint32_t));
    b = (uint32_t*)malloc(N * sizeof(uint32_t));
    // fin_out = (align_m*)malloc(NUM_BLOCKS * sizeof(align_m));

    //将double数据转换为posit格式并存入ab中
    for(int i=0;i<N;i++){
        posit = convertDoubleToPosit(wgts_data[i],in_BITS,ES);
        a[i] = posit.v;
        posit = convertDoubleToPosit(acts_data[i],in_BITS,ES);
        b[i] = posit.v;
        // a[i] = 0x3000;
        // b[i] = 0x2000;
    }
    //原矩阵输出
    posit = convertDoubleToPosit(outs_data,out_BITS,ES);
    printf("%x\n",posit.v);
    printf("%lf\n",outs_data);
    
    posit32_t posit_out;
    uint32_t acc[1] = {0};
    // // //调用PDPU
    // posit_out.v = cuPDPU_16(a,b)<<16;
    // //PDPU输出
    // printf("%x\n",posit_out.v);
    // printf("%lf\n",convertPositToDouble(posit_out,16,ES));

    
    posit_out.v = mac_dpu(a,b,acc);
    printf("%x\n",posit_out.v);
    printf("%lf\n",convertPositToDouble(posit_out,out_BITS,ES));

    // for(int n = NUM_BLOCKS;;n = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK){
    //     fin_add<<<(n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(dout,dfin_out,n);
    //     if(n < THREADS_PER_BLOCK) break;
    //     else{
    //         dout = dfin_out;
    //     }
    // }
    // hipDeviceSynchronize();
    
    // //test
    // hipMemcpy(fin_out, dfin_out, NUM_BLOCKS * sizeof(align_m), hipMemcpyDeviceToHost);

    // printf("%llx\n",((int64_t)(fin_out[0].m[0])));//>>(62-ALIGN_BIT));
    // printf("%llx\n",fin_out[0].m[1]);
    // printf("%llx\n",fin_out[0].m[2]);
    // printf("%llx\n",fin_out[0].m[3]);
    return 0;
}

// int main(){
//     //变量定义
//     uint32_t* test;
//     uint64_t* c_out;
//     // test[0] = 0x5000;
//     // test[1] = 0xc000;
//     // test[2] = 0x4000;

//     // posit16_mac(test,test+1,test+2,test+3);
//     // cout << "out = 0x" << hex << test[3] << endl;

//     //  uint16_t* d_test;
//     //内存分配
//     test = (uint32_t*)malloc(2*sizeof(uint32_t));
//     c_out = (uint64_t*)malloc(2*sizeof(uint64_t));
//     // hipMalloc(&d_test,4*sizeof(uint16_t));
//     //主机变量赋值
//     test[0] = 0x58000000;
//     test[1] = 0x40000000;
//     c_out[0] = 0x40000000;

//     // uint64_t decode[6]={0,0,0x80000000,0,0,0};
//     //内存复制到设备
//     // hipMemcpy(d_test ,test ,4*sizeof(uint16_t) , hipMemcpyHostToDevice);
//     //调用函数
//     // posit16_mac(d_test,d_test+1,d_test+2,d_test+3,0,0);
//     posit_mac(test,test+1,c_out,c_out+1,0,0);
//     cout << "test = " << hex << c_out[1] << endl;
//     // posit_mac(test,test+1,decode,test+3,1,0);
//     // cout << "test = " << hex << test[3] << endl;
//     //内存复制到主机
//     // hipMemcpy(test,d_test, 4*sizeof(uint16_t), hipMemcpyDeviceToHost);
//     //输出
    
//     // for(int i = 0;i<6;i++){
//     //     cout << "decode = " << hex << decode[i] << endl;
//     // }
//     return 0;

//     // Posit16 a;
//     // a.init(0x5800);
//     // a.decode();
//     // a.pprint('e');
//     // a.pprint('m');
//     // cout<<"\n"<<a.s<<endl;
//     return 0;
// }

int main_test_mac(){
    uint32_t test_ab[2] = {convertDoubleToPosit(0.0135,in_BITS,ES).v,convertDoubleToPosit(2.0335,in_BITS,ES).v};
    uint64_t test_co[2] = {convertDoubleToPosit(8,in_BITS,ES).v,0};
    // uint64_t test_co[2] = {0x67000000,0};

    // uint64_t decode[6] = {0,0,0x000000000000000,0,0,0};
    cout<<hex<<"a:"<<test_ab[0]<<"\nb:"<<test_ab[1]<<"\nc:"<<test_co[0]<<endl;
    posit_mac(test_ab,test_ab+1,test_co,test_co+1,0,0);
    cout<<hex<<test_co[1]<<endl;
    // posit_mac(test_ab,test_ab+1,decode,test_co+1,1,0);
    // cout<<hex<<test_co[1]<<endl;
    return 0;
}

//test decode&encode
int main_de_encode(){
    // Posit A;
    // // posit32_t p = convertDoubleToPosit(0.5,16,ES);
    // A.init(0x58001000);
    // cout<<"posit:"<<hex<<A.posit<<endl;
    // A.decode();
    // cout<<"e:"<<dec<<A.e<<"\n"<<"m:"<<hex<<A.m<<endl;
    // A.encode(A.s,A.e,A.m);
    // cout<<"posit:"<<hex<<A.posit<<endl;
    frac_align F,F1;
    F.init(0xffff0000ffff0000);
    F1.init(0x0000ffff0000ffff);
    F.shift(1);
    cout<<hex<<F.m[0]<<endl;
    F.shift(-2);
    cout<<hex<<F.m[0]<<endl;
    F.shift(1);
    cout<<hex<<F.m[0]<<endl;
    F.complement();
    cout<<hex<<F.m[0]<<endl;
    F.complement();
    cout<<hex<<F.m[0]<<endl;
    F.add(F1.m);
    cout<<hex<<F.m[0]<<endl;
    return 0;
}

// int main(){
//     main_test_mac();
//     return 0 ;
// }