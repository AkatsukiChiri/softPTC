#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "include/posit.cuh"
#include "include/cupdpu.cuh"
#include <math.h>
#include"SoftPositE.h"

__global__ void a_mul_b_device(uint32_t* a, uint32_t* b, uint64_t* out){
    // __shared__ uint64_t* matrix[16] = {0};
    int tid = threadIdx.x;
    int x = tid/4, y = tid%4;
    for(int k=0;k<4;k++){
        if(k==0) out[x*4+y] = 0;
        posit_mac(a+x*4+k,b+k*4+y,out+x*4+y,out+x*4+y,0,0);
        __syncthreads();
    }
    return;
}

__global__ void mul_add_device(uint32_t* a,uint32_t* b,uint64_t* c,uint64_t* out){
    int tid = threadIdx.x;
    int tid_x = tid/4 , tid_y = tid%4;
    int blk_x = blockIdx.x/(NJ/4) , blk_y = blockIdx.x%(NJ/4);
    int x = blk_x*4+tid_x , y = blk_y*4+tid_y;
    uint64_t decode[6] = {0};

    // __shared__ uint64_t out_perPTC[16*NI/4*NJ/4*NK/4] = {0};

    *(out+x*NJ+y) = *(c+x*NJ+y);
    for(int k=0;k<(NK/4);k++){
        posit_mac(a+x*NK+k*4+0,b+(0+k*4)*NJ+y,out+x*NJ+y,decode,0,1);
        posit_mac(a+x*NK+k*4+1,b+(1+k*4)*NJ+y,decode,decode,1,1);
        posit_mac(a+x*NK+k*4+2,b+(2+k*4)*NJ+y,decode,decode,1,1);
        posit_mac(a+x*NK+k*4+3,b+(3+k*4)*NJ+y,decode,out+x*NJ+y,1,0);

        // posit_mac(a+x*NK+k*4+0,b+(0+k*4)*NJ+y,out+x*NJ+y,out+x*NJ+y,0,0);
        // posit_mac(a+x*NK+k*4+1,b+(1+k*4)*NJ+y,out+x*NJ+y,out+x*NJ+y,0,0);
        // posit_mac(a+x*NK+k*4+2,b+(2+k*4)*NJ+y,out+x*NJ+y,out+x*NJ+y,0,0);
        // posit_mac(a+x*NK+k*4+3,b+(3+k*4)*NJ+y,out+x*NJ+y,out+x*NJ+y,0,0);
    }
    
    return;
}

void PTC(uint32_t* a, uint32_t* b, uint32_t* acc, uint32_t* PTC_out){
    uint64_t* acc_host,* PTC_out_host;
    acc_host = (uint64_t*)malloc(NI*NJ*sizeof(uint64_t));
    PTC_out_host = (uint64_t*)malloc(NI*NJ*sizeof(uint64_t));
    //init acc_host
    for(int i=0;i<NI;i++){
        for(int j=0;j<NJ;j++){
            acc_host[i*NI+j] = acc[i*NI+j];
        }
    }

    uint32_t * d_a,* d_b;
    uint64_t * d_acc,* d_out;
    hipMalloc(&d_a,NI*NK*sizeof(uint32_t));
    hipMalloc(&d_b,NJ*NK*sizeof(uint32_t));
    hipMalloc(&d_acc,NI*NJ*sizeof(uint64_t));
    hipMalloc(&d_out,NI*NJ*sizeof(uint64_t));

    hipMemcpy(d_a,a,NI*NK*sizeof(uint32_t),hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,NJ*NK*sizeof(uint32_t),hipMemcpyHostToDevice);
    hipMemcpy(d_acc,acc_host,NI*NJ*sizeof(uint64_t),hipMemcpyHostToDevice);

    mul_add_device<<<NI*NJ/16,16>>>(d_a,d_b,d_acc,d_out);

    hipMemcpy(PTC_out_host,d_out,NI*NJ*sizeof(uint64_t),hipMemcpyDeviceToHost);

    // matrix_print<uint64_t>(PTC_out_host);
    // cout<<hex<<PTC_out_host[8]<<endl;

    for(int i=0;i<NI;i++){
        for(int j=0;j<NJ;j++){
            PTC_out[i*NI+j] = PTC_out_host[i*NI+j];
        }
    }
    return;
}

void PTC_im(posit_im* a, posit_im* b, posit_im* acc, posit_im* PTC_out, int length){
    uint32_t* A, *B, *ACC, *OUT;
    A = (uint32_t*)malloc(4*length*length*sizeof(uint32_t));
    B = (uint32_t*)malloc(4*length*length*sizeof(uint32_t));
    ACC = (uint32_t*)malloc(4*length*length*sizeof(uint32_t));
    OUT = (uint32_t*)malloc(4*length*length*sizeof(uint32_t));
    //init
    for(int i=0;i<2*length;i++){
        for(int j=0;j<2*length;j++){
            A[i*2*length+j]=0;
            B[i*2*length+j]=0;
            ACC[i*2*length+j]=0;
            OUT[i*2*length+j]=0;
            if(i%2==0)
                if(j%2==0){
                    A[i*2*length+j]=a[i/2*length+j/2].r;
                    B[i*2*length+j]=b[i/2*length+j/2].r;
                    ACC[i*2*length+j]=acc[i/2*length+j/2].r;
                }
                else{
                    A[i*2*length+j]=-a[i/2*length+j/2].i;
                    B[i*2*length+j]=b[i/2*length+j/2].i;
                }
            else
                if(j%2==0){
                    A[i*2*length+j]=a[i/2*length+j/2].r;
                    B[i*2*length+j]=b[i/2*length+j/2].i;
                }
                else{
                    A[i*2*length+j]=a[i/2*length+j/2].i;
                    B[i*2*length+j]=b[i/2*length+j/2].r;
                    ACC[i*2*length+j]=acc[i/2*length+j/2].i;
                }
        }
    }

    // printf("A:\n");
    // matrix_print<uint32_t>(A);
    // printf("B:\n");
    // matrix_print<uint32_t>(B);

    PTC(A,B,ACC,OUT);
    for(int i=0;i<2*length;i++){
        for(int j=0;j<2*length;j++){
            if((i%2==0)&&(j%2==0)) PTC_out[i/2*length+j/2].r=OUT[i*2*length+j];
            else if((i%2!=0)&&(j%2!=0)) PTC_out[i/2*length+j/2].i=OUT[i*2*length+j];
        }
    }
    return;
}

void FFT(uint32_t* x, int length, uint32_t* out_r, uint32_t* out_i){
    posit_im W[length*length] = {0};
    posit_im X[length*length] = {0};
    posit_im OUT[length*length] = {0};
    for(int i=0;i<length;i++){
        for(int j=0;j<length;j++){
            if(j==0) {X[i*length].r=x[i];X[i*length].i=0;}
            W[i*length+j].r = (convertDoubleToPosit(cos(-2*3.1415626*i*j/length),in_BITS,ES)).v;
            W[i*length+j].i = (convertDoubleToPosit(sin(-2*3.1415626*i*j/length),in_BITS,ES)).v;
            OUT[i*length+j].r = 0;
            OUT[i*length+j].i = 0;
        }
    }
    PTC_im(W,X,OUT,OUT,length);
    for(int i=0;i<length;i++){out_r[i]=OUT[i*length].r;out_i[i]=OUT[i*length].i;}
    return;
}