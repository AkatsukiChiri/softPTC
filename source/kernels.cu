#include "hip/hip_runtime.h"
#include "include/posit.cuh"
#include "include/cupdpu.cuh"

__global__ void a_mul_b_device(uint32_t* a, uint32_t* b, uint64_t* out){
    // __shared__ uint64_t* matrix[16] = {0};
    int tid = threadIdx.x;
    int x = tid/4, y = tid%4;
    for(int k=0;k<4;k++){
        if(k==0) out[x*4+y] = 0;
        posit_mac(a+x*4+k,b+k*4+y,out+x*4+y,out+x*4+y,0,0);
        __syncthreads();
    }
    return;
}