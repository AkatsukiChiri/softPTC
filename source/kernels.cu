#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "include/posit.cuh"
#include "include/cupdpu.cuh"

__global__ void a_mul_b_device(uint32_t* a, uint32_t* b, uint64_t* out){
    // __shared__ uint64_t* matrix[16] = {0};
    int tid = threadIdx.x;
    int x = tid/4, y = tid%4;
    for(int k=0;k<4;k++){
        if(k==0) out[x*4+y] = 0;
        posit_mac(a+x*4+k,b+k*4+y,out+x*4+y,out+x*4+y,0,0);
        __syncthreads();
    }
    return;
}

__global__ void mul_add_device(uint32_t* a,uint32_t* b,uint64_t* c,uint64_t* out){
    int tid = threadIdx.x;
    int tid_x = tid/4 , tid_y = tid%4;
    int blk_x = blockIdx.x/(NJ/4) , blk_y = blockIdx.x%(NJ/4);
    int x = blk_x*4+tid_x , y = blk_y*4+tid_y;
    uint64_t decode[6] = {0};

    // __shared__ uint64_t out_perPTC[16*NI/4*NJ/4*NK/4] = {0};

    *(out+x*NJ+y) = *(c+x*NJ+y);
    for(int k=0;k<(NK/4);k++){
        posit_mac(a+x*NJ+k*4+0,b+(0+k*4)*NJ+y,out+x*NJ+y,decode,0,1);
        posit_mac(a+x*NJ+k*4+1,b+(1+k*4)*NJ+y,decode,decode,1,1);
        posit_mac(a+x*NJ+k*4+2,b+(2+k*4)*NJ+y,decode,decode,1,1);
        posit_mac(a+x*NJ+k*4+3,b+(3+k*4)*NJ+y,decode,out+x*NJ+y,1,0);
    }
    
    return;
}

void PTC(uint32_t* a, uint32_t* b, uint32_t* acc, uint32_t* PTC_out){
    uint64_t* acc_host,* PTC_out_host;
    acc_host = (uint64_t*)malloc(NI*NJ*sizeof(uint64_t));
    PTC_out_host = (uint64_t*)malloc(NI*NJ*sizeof(uint64_t));
    //init acc_host
    for(int i=0;i<NI;i++){
        for(int j=0;j<NJ;j++){
            acc_host[i*NI+j] = acc[i*NI+j];
        }
    }

    uint32_t * d_a,* d_b;
    uint64_t * d_acc,* d_out;
    hipMalloc(&d_a,NI*NK*sizeof(uint32_t));
    hipMalloc(&d_b,NJ*NK*sizeof(uint32_t));
    hipMalloc(&d_acc,NI*NJ*sizeof(uint64_t));
    hipMalloc(&d_out,NI*NJ*sizeof(uint64_t));

    hipMemcpy(d_a,a,NI*NK*sizeof(uint32_t),hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,NJ*NK*sizeof(uint32_t),hipMemcpyHostToDevice);
    hipMemcpy(d_acc,acc_host,NI*NJ*sizeof(uint64_t),hipMemcpyHostToDevice);

    mul_add_device<<<NI*NJ/16,16>>>(d_a,d_b,d_acc,d_out);

    hipMemcpy(PTC_out_host,d_out,NI*NJ*sizeof(uint64_t),hipMemcpyDeviceToHost);

    // matrix_print<uint64_t>(PTC_out_host);
    // cout<<hex<<PTC_out_host[8]<<endl;

    for(int i=0;i<NI;i++){
        for(int j=0;j<NJ;j++){
            PTC_out[i*NI+j] = PTC_out_host[i*NI+j];
        }
    }
    return;
}