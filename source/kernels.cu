#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "include/posit.cuh"
#include "include/cupdpu.cuh"

__global__ void a_mul_b_device(uint32_t* a, uint32_t* b, uint64_t* out){
    // __shared__ uint64_t* matrix[16] = {0};
    int tid = threadIdx.x;
    int x = tid/4, y = tid%4;
    for(int k=0;k<4;k++){
        if(k==0) out[x*4+y] = 0;
        posit_mac(a+x*4+k,b+k*4+y,out+x*4+y,out+x*4+y,0,0);
        __syncthreads();
    }
    return;
}

__global__ void mul_add_device(uint32_t* a,uint32_t* b,uint64_t* c,uint64_t* out){
    int tid = threadIdx.x;
    int x = tid/4 , y = tid%4;
    uint64_t decode[6] = {0};

    __shared__ uint64_t out_perPTC[16*NI/4*NJ/4*NK/4];

    *(out+x*4+y) = 0;
    posit_mac(a+x*4+0,b+0*4+y,c+x*4+y,decode,0,1);
    posit_mac(a+x*4+1,b+1*4+y,decode,decode,1,1);
    posit_mac(a+x*4+2,b+2*4+y,decode,decode,1,1);
    posit_mac(a+x*4+3,b+3*4+y,decode,out+x*4+y,1,0);
    return;
}

void PTC(uint32_t* a, uint32_t* b, uint32_t* acc, uint32_t* PTC_out){
    uint64_t* acc_host,* PTC_out_host;
    acc_host = (uint64_t*)malloc(NI*NJ*sizeof(uint64_t));
    PTC_out_host = (uint64_t*)malloc(NI*NJ*sizeof(uint64_t));
    //init acc_host
    for(int i=0;i<NI;i++){
        for(int j=0;j<NJ;j++){
            acc_host[i*NI+j] = acc[i*NI+j];
        }
    }

    uint32_t * d_a,* d_b;
    uint64_t * d_acc,* d_out;
    hipMalloc(&d_a,NI*NK*sizeof(uint32_t));
    hipMalloc(&d_b,NJ*NK*sizeof(uint32_t));
    hipMalloc(&d_acc,NI*NJ*sizeof(uint64_t));
    hipMalloc(&d_out,NI*NJ*sizeof(uint64_t));

    hipMemcpy(d_a,a,NI*NK*sizeof(uint32_t),hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,NJ*NK*sizeof(uint32_t),hipMemcpyHostToDevice);
    hipMemcpy(d_acc,acc_host,NI*NJ*sizeof(uint64_t),hipMemcpyHostToDevice);

    mul_add_device<<<1,16>>>(d_a,d_b,d_acc,d_out);

    hipMemcpy(PTC_out_host,d_out,NI*NJ*sizeof(uint64_t),hipMemcpyDeviceToHost);

    // matrix_print<uint64_t>(PTC_out_host);
    // cout<<hex<<PTC_out_host[8]<<endl;

    for(int i=0;i<NI;i++){
        for(int j=0;j<NJ;j++){
            PTC_out[i*NI+j] = PTC_out_host[i*NI+j];
        }
    }
    return;
}