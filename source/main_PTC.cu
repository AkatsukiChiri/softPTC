#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

#include"include/cupdpu.cuh"
#include"include/double2posit.cuh"
#include"include/posit.cuh"
#include"include/rw_bin.cuh"
#include"include/cupdpu.cuh"
#include"SoftPositE.h"

using namespace std;

void init(double *a,double *b,double *c){
    for(int i=0;i<NI*NK;i++) a[i]=(rand()%100000)/20000.0-2.5;
    for(int i=0;i<NJ*NK;i++) b[i]=(rand()%100000)/20000.0-2.5;
    for(int i=0;i<NI*NJ;i++) c[i]=(rand()%100000)/20000.0-2.5;
}

void gemm(double *a,double *b,double *c,double *out){
    for(int i=0;i<NI;i++){
        for(int j=0;j<NJ;j++){
            out[i*NJ+j] = c[i*NJ+j];
            for(int k=0;k<NK;k++){
                out[i*NJ+j] += a[i*NK+k] * b[k*NJ+j];
            }
        }
    }
}

void gemm_fp(double *a,double *b,double *c,double *out){
    float fp_a,fp_b,fp_c;double fp_out;
    for(int i=0;i<NI;i++){
        for(int j=0;j<NJ;j++){
            fp_c = c[i*NJ+j];
            fp_out = fp_c;
            for(int k=0;k<NK;k++){
                fp_a = a[i*NK+k];
                fp_b = b[k*NJ+j];
                fp_out += fp_a * fp_b;
            }
            out[i*NJ+j] = fp_out;
        }
    }
    return;
}

double MSE(double* out,double* out_double){
    double err = 0;
    for(int i=0;i<NI*NJ;i++) err += (out[i]-out_double[i])*(out[i]-out_double[i]);
    err /= (NI*NJ);
    return err;
}

double detect(double* out,double* out_double,double threshold){
    double count = 0;
    for(int i=0;i<NI*NJ;i++) if(abs((out[i]-out_double[i])/out_double[i])>threshold) count+=1;
    return count/(NI*NJ);
}

void main_ptc_test(){
    double a[NI*NK]   = {0};
    double b[NJ*NK]   = {0}; 
    double c[NI*NJ]   = {0};
    double out[NI*NJ] = {0};

    init(a,b,c);

    uint32_t A[NI*NK] = {0};
    uint32_t B[NJ*NK] = {0};
    uint32_t C[NI*NJ] = {0};
    uint32_t OUT[NI*NJ] = {0};

    for(int i=0;i<NI*NK;i++) A[i] = (convertDoubleToPosit(a[i],in_BITS,ES)).v;
    for(int i=0;i<NJ*NK;i++) B[i] = (convertDoubleToPosit(b[i],in_BITS,ES)).v;
    for(int i=0;i<NI*NJ;i++) C[i] = (convertDoubleToPosit(c[i],in_BITS,ES)).v;

    PTC(A,B,C,OUT);

    posit32_t p;
    for(int i=0;i<NI*NJ;i++) {
        p.v = OUT[i];
        out[i] = convertPositToDouble(p,out_BITS,ES);
        }
    
    double out_double[NI*NJ] = {0};
    double out_fp[NI*NJ] = {0};
    gemm(a,b,c,out_double);

    gemm_fp(a,b,c,out_fp);
    
    cout<<"the error of Posit in MSE is : "<<MSE(out,out_double)<<endl;
    cout<<"the error of FP in MSE is : "<<MSE(out_fp,out_double)<<endl;
    cout<<"the ratio of relative error exceeds 0.05: "<<detect(out,out_double,0.05)<<endl;
    cout<<"the ratio of relative error exceeds 0.1: "<<detect(out,out_double,0.1)<<endl;
    cout<<hex<<OUT[1]<<endl;
    // printf("\nA:\n");
    // matrix_print<double>(a);  
    // printf("\nB:\n");
    // matrix_print<double>(b);  
    // printf("\nC:\n");
    // matrix_print<double>(c);  
    // printf("\nout\n");
    // matrix_print<double>(out);    
    // printf("\n");
    // matrix_print<double>(out_double);
    // if(err) printf("the error of MSE is : %lf",err);
    
    // else printf("the same!!");
}

int main(){
    main_ptc_test();
}