#include<iostream>
#include<hip/hip_runtime.h>

#include"include/cupdpu.cuh"
#include"include/double2posit.cuh"
#include"include/posit.cuh"
#include"include/rw_bin.cuh"
#include"include/cupdpu.cuh"
#include"SoftPositE.h"

using namespace std;

void main_ptc_test(){
    double a[NI*NK] = { 1,0,0,0,
                        0,0.8,0,0,
                        0,0,7,0,
                        0,0,0,0.25,};
    double b[NJ*NK] = { 1,0,0,0,
                        0,3,0,0,
                        0,0,0.1,0,
                        0,0,0,9,}; 
    double c[NI*NJ] = { 0,0,2,0,
                        0,0,0,0.3,
                        0,5,0,0,
                        0,0,0,0,};  
    double out[NI*NJ]={ 0       };

    uint32_t A[NI*NK] = {0};
    uint32_t B[NJ*NK] = {0};
    uint32_t C[NI*NJ] = {0};
    uint32_t OUT[16] = {0};

    for(int i=0;i<NI*NK;i++) A[i] = (convertDoubleToPosit(a[i],in_BITS,ES)).v;
    for(int i=0;i<NJ*NK;i++) B[i] = (convertDoubleToPosit(b[i],in_BITS,ES)).v;
    for(int i=0;i<NI*NJ;i++) C[i] = (convertDoubleToPosit(c[i],in_BITS,ES)).v;

    PTC(A,B,C,OUT);

    posit32_t p;
    for(int i=0;i<NI*NJ;i++) {
        p.v = OUT[i];
        out[i] = convertPositToDouble(p,out_BITS,ES);
        }

    matrix_print<double>(out);                                       
}

int main(){
    main_ptc_test();
}