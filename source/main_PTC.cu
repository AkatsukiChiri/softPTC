#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_fp16.h>

#include"include/cupdpu.cuh"
#include"include/double2posit.cuh"
#include"include/posit.cuh"
#include"include/rw_bin.cuh"
#include"include/cupdpu.cuh"
#include"SoftPositE.h"

using namespace std;

void init(double *a,double *b,double *c){
    for(int i=0;i<NI*NK;i++) a[i]=(rand()%100000)/20000.0-2.5;
    for(int i=0;i<NJ*NK;i++) b[i]=(rand()%100000)/20000.0-2.5;
    for(int i=0;i<NI*NJ;i++) c[i]=(rand()%100000)/20000.0-2.5;
}

void gemm(double *a,double *b,double *c,double *out){
    for(int i=0;i<NI;i++){
        for(int j=0;j<NJ;j++){
            out[i*NJ+j] = c[i*NJ+j];
            for(int k=0;k<NK;k++){
                out[i*NJ+j] += a[i*NK+k] * b[k*NJ+j];
            }
        }
    }
}

__global__ void gemm_fp_D(double *a,double *b,double *c,double *out){
    float fp_a,fp_b,fp_c;double fp_ab,fp_out;
    for(int i=0;i<NI;i++){
        for(int j=0;j<NJ;j++){
            fp_c = c[i*NJ+j];
            fp_out = fp_c;
            for(int k=0;k<NK;k++){
                fp_a = a[i*NK+k];
                fp_b = b[k*NJ+j];
                fp_ab = fp_a*fp_b;
                fp_out += fp_ab;
            }
            out[i*NJ+j] = fp_out;
        }
    }
    return;
}

void gemm_fp(double *a,double *b,double *c,double *out){
    double *d_a,*d_b,*d_c,*d_out;
    hipMalloc(&d_a,NI*NK*sizeof(double));
    hipMalloc(&d_b,NJ*NK*sizeof(double));
    hipMalloc(&d_c,NI*NJ*sizeof(double));
    hipMalloc(&d_out,NI*NJ*sizeof(double));

    hipMemcpy(d_a,a,NI*NK*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,NJ*NK*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_c,c,NI*NJ*sizeof(double),hipMemcpyHostToDevice);

    gemm_fp_D<<<1,1>>>(d_a,d_b,d_c,d_out);

    hipMemcpy(out,d_out,NI*NJ*sizeof(double),hipMemcpyDeviceToHost);
}

double MSE(double* out,double* out_double){
    double err = 0;
    for(int i=0;i<NI*NJ;i++) err += (out[i]-out_double[i])*(out[i]-out_double[i]);
    err /= (NI*NJ);
    return err;
}

double detect(double* out,double* out_double,double threshold){
    double count = 0;
    for(int i=0;i<NI*NJ;i++) if(abs((out[i]-out_double[i])/out_double[i])>threshold) count+=1;
    return count/(NI*NJ);
}

void main_ptc_test(){
    double a[NI*NK]   = {0};
    double b[NJ*NK]   = {0}; 
    double c[NI*NJ]   = {0};
    double out[NI*NJ] = {0};

    init(a,b,c);

    uint32_t A[NI*NK] = {0};
    uint32_t B[NJ*NK] = {0};
    uint32_t C[NI*NJ] = {0};
    uint32_t OUT[NI*NJ] = {0};

    for(int i=0;i<NI*NK;i++) A[i] = (convertDoubleToPosit(a[i],in_BITS,ES)).v;
    for(int i=0;i<NJ*NK;i++) B[i] = (convertDoubleToPosit(b[i],in_BITS,ES)).v;
    for(int i=0;i<NI*NJ;i++) C[i] = (convertDoubleToPosit(c[i],in_BITS,ES)).v;

    PTC(A,B,C,OUT);

    posit32_t p;
    for(int i=0;i<NI*NJ;i++) {
        p.v = OUT[i];
        out[i] = convertPositToDouble(p,out_BITS,ES);
        }
    
    double out_double[NI*NJ] = {0};
    double out_fp[NI*NJ] = {0};
    gemm(a,b,c,out_double);

    gemm_fp(a,b,c,out_fp);
    
    cout<<"the error of Posit in MSE is : "<<MSE(out,out_double)<<endl;
    cout<<"the error of FP in MSE is : "<<MSE(out_fp,out_double)<<endl;
    cout<<"the ratio of relative error exceeds 0.05: "<<detect(out,out_double,0.05)<<endl;
    cout<<"the ratio of relative error exceeds 0.1: "<<detect(out,out_double,0.1)<<endl;
    cout<<hex<<OUT[1]<<endl;
    // printf("\nA:\n");
    // matrix_print<double>(a);  
    // printf("\nB:\n");
    // matrix_print<double>(b);  
    // printf("\nC:\n");
    // matrix_print<double>(c);  
    // printf("\nout\n");
    // matrix_print<double>(out);    
    // printf("\n");
    // matrix_print<double>(out_double);
    // if(err) printf("the error of MSE is : %lf",err);
    
    // else printf("the same!!");
}

void main_PTCim_test()
{
    int length = 4;
    posit_im a[length*length] = {0};
    posit_im b[length*length]  = {0}; 
    posit_im c[length*length]  = {0};
    posit_im out[length*length] = {0};
    for(int i=0;i<length*length;i++){
        a[i].r=(convertDoubleToPosit(1,in_BITS,ES)).v;a[i].i=(convertDoubleToPosit(1,in_BITS,ES)).v;
        b[i].r=(convertDoubleToPosit(1,in_BITS,ES)).v;b[i].i=(convertDoubleToPosit(-1,in_BITS,ES)).v;
        c[i].r=(convertDoubleToPosit(0,in_BITS,ES)).v;c[i].i=(convertDoubleToPosit(0,in_BITS,ES)).v;
        out[i].r=(convertDoubleToPosit(0,in_BITS,ES)).v;out[i].i=(convertDoubleToPosit(0,in_BITS,ES)).v;
    }
    PTC_im(a,b,c,out,length);
    printf("%x",out[0].r);
}

void main_FFT(){
    int length = 16;
    double x[length] = {0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15};
    uint32_t p_x[length]={0};
    double out_r[length]={0},out_i[length]={0};
    uint32_t p_out_r[length]={0},p_out_i[length]={0};
    for(int i=0;i<length;i++) p_x[i] = (convertDoubleToPosit(x[i],in_BITS,ES)).v;
    FFT(p_x,length,p_out_r,p_out_i);
    posit32_t p;
    for(int i=0;i<length;i++) {
        p.v = p_out_r[i];
        out_r[i] = convertPositToDouble(p,out_BITS,ES);
        p.v = p_out_i[i];
        out_i[i] = convertPositToDouble(p,out_BITS,ES);
    }
    for(int i=0;i<length;i++) printf("%lf + %lfi\n",out_r[i],out_i[i]);
}

int main(){
    main_FFT();
}