#include "include/posit.cuh"
#include "include/cupdpu.cuh"

void PositTensorCore::init(uint32_t *A,uint32_t *B,uint64_t *C){
    for(int i=0;i<16;i++){
        a[i] = A[i];
        b[i] = B[i];
        c[i] = C[i];
        out[i] = 0;
    }
    return;
}

void PositTensorCore::a_mul_b(){
    uint32_t *d_a,*d_b;uint64_t *d_out;
    hipMalloc(&d_a,16*sizeof(uint32_t));
    hipMalloc(&d_b,16*sizeof(uint32_t));
    hipMalloc(&d_out,16*sizeof(uint64_t));
    hipMemcpy(d_a, a, 16*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, 16*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, 16*sizeof(uint64_t), hipMemcpyHostToDevice);

    a_mul_b_device<<<1,16>>>(d_a,d_b,d_out);

    hipMemcpy(out, d_out, 16*sizeof(uint64_t), hipMemcpyDeviceToHost);
    return;
}

void PositTensorCore::mul_add(){
    a_mul_b();
    uint32_t one[1] = {0x40000000};
    for(int i=0;i<4;i++){
        for(int j=0;j<4;j++){
            uint32_t temp[1] = {(uint32_t)out[i*4+j]};
            posit_mac(temp,one,c+i*4+j,out+i*4+j,0,0);
        }
    }
}

