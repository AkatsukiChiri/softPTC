#include "hip/hip_runtime.h"
#include<math.h>

#include"include/cupdpu.cuh"
#include"include/double2posit.cuh"
#include"SoftPositE.h"

uint16_t cuPDPU_16(uint16_t * a,uint16_t * b){
    align_m* out;
    uint16_t *da,*db;
    align_m *dout;

    out = (align_m*)malloc(NUM_BLOCKS * sizeof(align_m));
    hipMalloc(&da, N * sizeof(uint16_t));
    hipMalloc(&db, N * sizeof(uint16_t));
    hipMalloc(&dout, NUM_BLOCKS * sizeof(align_m));

    hipMemcpy(da, a, N * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(db, b, N * sizeof(uint16_t), hipMemcpyHostToDevice);

    kernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(da,db,dout);

    hipMemcpy(out, dout, NUM_BLOCKS * sizeof(align_m), hipMemcpyDeviceToHost);

    double double_out = (int64_t)out[0].m[0]/(double)pow(2,62-ALIGN_BIT) + out[0].m[1]/(double)pow(2,64 + 62-ALIGN_BIT);
    posit32_t posit = convertDoubleToPosit(double_out,16,ES);
    return posit.v>>16;
}

__global__ void kernel(uint16_t *a, uint16_t *b, align_m* out)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ align_m m_ab_array[THREADS_PER_BLOCK];
    if(tid>=N){
        m_ab_array[threadIdx.x].m[0]=0;
        m_ab_array[threadIdx.x].m[1]=0;
        m_ab_array[threadIdx.x].m[2]=0;
        m_ab_array[threadIdx.x].m[3]=0;
        return;
    }
    else{
        //deposit
        posit pa, pb;
        pa = deposit(a[tid]);
        pb = deposit(b[tid]);
        //get ma mb
        uint16_t ma = pa.m;
        uint16_t mb = pb.m;
        //get e_ab
        __shared__ int e_ab_array[THREADS_PER_BLOCK];
        e_ab_array[threadIdx.x] =((pa.e!=0x80000000)&&(pb.e!=0x80000000))?pa.e + pb.e:0x80000000;

        __syncthreads();
        //find e_max in block
        int e_max = 0x80000000;
        for(int i=0;i < THREADS_PER_BLOCK;i++){
            if(e_max < e_ab_array[i]) e_max = e_ab_array[i];
        }
        //align in block
        /*
        __shared__ uint32_t m_ab_array[THREADS_PER_BLOCK];
        m_ab_array[threadIdx.x] = ma * mb;
        m_ab_array[threadIdx.x] = m_ab_array[threadIdx.x] >> (e_max - e_ab_array[threadIdx.x]);
        if(pa.s^pb.s) m_ab_array[threadIdx.x] = ~m_ab_array[threadIdx.x] + 1;
        */
        
        m_ab_array[threadIdx.x].m[0] = (uint64_t)(ma * mb) << 32;
        m_ab_array[threadIdx.x].m[1] = 0;
        m_ab_array[threadIdx.x].m[2] = 0;
        m_ab_array[threadIdx.x].m[3] = 0;

        m_ab_array[threadIdx.x].m[1] = (64 - (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT))>0 ? m_ab_array[threadIdx.x].m[0] << (64 - (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT)) : m_ab_array[threadIdx.x].m[0] >> -(64 - (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT));
        m_ab_array[threadIdx.x].m[2] = (128 - (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT))>0 ? m_ab_array[threadIdx.x].m[0] << (128 - (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT)) : m_ab_array[threadIdx.x].m[0] >> -(128 - (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT));
        m_ab_array[threadIdx.x].m[3] = (192 - (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT))>0 ? m_ab_array[threadIdx.x].m[0] << (192 - (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT)) : m_ab_array[threadIdx.x].m[0] >> -(192 - (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT));
        m_ab_array[threadIdx.x].m[0] = (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT)>0 ? m_ab_array[threadIdx.x].m[0] >> (e_max - e_ab_array[threadIdx.x] + ALIGN_BIT) : m_ab_array[threadIdx.x].m[0] << (-(e_max - e_ab_array[threadIdx.x] + ALIGN_BIT));
    
        //2的补码
        if(pa.s^pb.s) {
            m_ab_array[threadIdx.x].m[0] = ~m_ab_array[threadIdx.x].m[0];
            m_ab_array[threadIdx.x].m[1] = ~m_ab_array[threadIdx.x].m[1];
            m_ab_array[threadIdx.x].m[2] = ~m_ab_array[threadIdx.x].m[2];
            m_ab_array[threadIdx.x].m[3] = ~m_ab_array[threadIdx.x].m[3];
            if(m_ab_array[threadIdx.x].m[3]!=0xffffffffffffffff) m_ab_array[threadIdx.x].m[3] = m_ab_array[threadIdx.x].m[3]+1;
            else{
                m_ab_array[threadIdx.x].m[3] = m_ab_array[threadIdx.x].m[3] + 1;
                if(m_ab_array[threadIdx.x].m[2]!=0xffffffffffffffff) m_ab_array[threadIdx.x].m[2] = m_ab_array[threadIdx.x].m[2]+1;
                else{
                    m_ab_array[threadIdx.x].m[2] = m_ab_array[threadIdx.x].m[2] + 1;
                    if(m_ab_array[threadIdx.x].m[1]!=0xffffffffffffffff) m_ab_array[threadIdx.x].m[1] = m_ab_array[threadIdx.x].m[1]+1;
                else{
                    m_ab_array[threadIdx.x].m[1] = m_ab_array[threadIdx.x].m[1] + 1;
                    m_ab_array[threadIdx.x].m[0] = m_ab_array[threadIdx.x].m[0] + 1;
                }
                }
            }
        }
        
        
        //accumulate in block

        int if_carry[4] = {0,0,0,0};
        for(int stride = blockDim.x / 2;stride > 0;stride >>= 1){
            if (threadIdx.x < stride) {
                if_carry[0] = 0;
                if_carry[1] = 0;
                if_carry[2] = 0;
                if_carry[3] = 0;
                //m3
                if(m_ab_array[threadIdx.x + stride].m[3] > (uint64_t)0xffffffffffffffff - m_ab_array[threadIdx.x].m[3]) if_carry[3] = 1;
                m_ab_array[threadIdx.x].m[3] += m_ab_array[threadIdx.x + stride].m[3];
                //if m3 carry
                if(m_ab_array[threadIdx.x + stride].m[2] > (uint64_t)0xffffffffffffffff - if_carry[3]) if_carry[2] = 1;
                m_ab_array[threadIdx.x].m[2] += if_carry[3];
                //m2
                if(m_ab_array[threadIdx.x + stride].m[2] > (uint64_t)0xffffffffffffffff - m_ab_array[threadIdx.x].m[2]) if_carry[2] = 1;
                m_ab_array[threadIdx.x].m[2] += m_ab_array[threadIdx.x + stride].m[2];
                //if m2 carry
                if(m_ab_array[threadIdx.x + stride].m[1] > (uint64_t)0xffffffffffffffff - if_carry[2]) if_carry[1] = 1;
                m_ab_array[threadIdx.x].m[1] += if_carry[2];
                //m1
                if(m_ab_array[threadIdx.x + stride].m[1] > (uint64_t)0xffffffffffffffff - m_ab_array[threadIdx.x].m[1]) if_carry[1] = 1;
                m_ab_array[threadIdx.x].m[1] += m_ab_array[threadIdx.x + stride].m[1];
                //if m1 carry
                if(m_ab_array[threadIdx.x + stride].m[0] > (uint64_t)0xffffffffffffffff - if_carry[1]) if_carry[0] = 1;
                m_ab_array[threadIdx.x].m[0] += if_carry[1];
                if(m_ab_array[threadIdx.x + stride].m[0] > (uint64_t)0xffffffffffffffff - m_ab_array[threadIdx.x].m[0]) if_carry[0] = 1;
                m_ab_array[threadIdx.x].m[0] += m_ab_array[threadIdx.x + stride].m[0];
            }
            __syncthreads();
        }

        //accumulate between blocks
        
        if(threadIdx.x == 0) {
            m_ab_array[0].m[3] = (e_max > 0) ? (m_ab_array[0].m[3] << e_max) : (m_ab_array[0].m[2] << (64 + e_max)) + (m_ab_array[0].m[3] >> -e_max);
            m_ab_array[0].m[2] = (e_max > 0) ? (m_ab_array[0].m[2] << e_max) + (m_ab_array[0].m[3] >> (64 - e_max)) : (m_ab_array[0].m[1] << (64 + e_max)) + (m_ab_array[0].m[2] >> -e_max);
            m_ab_array[0].m[1] = (e_max > 0) ? (m_ab_array[0].m[1] << e_max) + (m_ab_array[0].m[2] >> (64 - e_max)) : (m_ab_array[0].m[0] << (64 + e_max)) + (m_ab_array[0].m[1] >> -e_max);
            m_ab_array[0].m[0] = (e_max > 0) ? (m_ab_array[0].m[0] << e_max) + (m_ab_array[0].m[1] >> (64 - e_max)) : (uint64_t)((int64_t)m_ab_array[0].m[0] >> (-e_max));
        }
        
        if(threadIdx.x==0){
            out[blockIdx.x].m[0] = m_ab_array[0].m[0];
            out[blockIdx.x].m[1] = m_ab_array[0].m[1];
            out[blockIdx.x].m[2] = m_ab_array[0].m[2];
            out[blockIdx.x].m[3] = m_ab_array[0].m[3];
        }
        return;
    }
    
    
}

__device__ __host__ posit deposit(uint16_t a){
    posit p;
    p.posit = a;
    if((a == 0x0000) || (a == 0x8000))
    {
        p.s = 0;
        p.k = 0;
        p.e = 0x80000000;
        p.m = 0;
        p.l_f = 0;
        return p;
    }
    p.s = a >> 15;
    if(p.s) a = (~a)+1;
    int ex=0;
    for(int i=0,temp=0x4000;i<=15;i++){
        if(((bool)(a & temp)) ^ ((bool)(a & (temp>>1)))){

            p.k = (a & temp) ? i : (-1-i);

            ex = a & ((temp>>1)-1);
            ex = ex >> (16-i-3-ES);

            break;
        }
        else temp=temp>>1;
    }
    //由k值得到e
    p.e=p.k*(2*ES)+ex;
    //求f的值
    p.l_f = (p.k<0) ? (14-ES+p.k) : (13-ES-p.k);
    p.m=a & ((0x0001<<p.l_f)-1);
    p.m = p.m << (15 - p.l_f);
    p.m = p.m | 0x8000;
    return p;
}

__global__ void fin_add(align_m *input,align_m *output,int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ align_m sdata[THREADS_PER_BLOCK];
    if(tid < size){
        sdata[threadIdx.x].m[0] = input[tid].m[0];
        sdata[threadIdx.x].m[1] = input[tid].m[1];
        sdata[threadIdx.x].m[2] = input[tid].m[2];
        sdata[threadIdx.x].m[3] = input[tid].m[3];
    }
    else{
        sdata[threadIdx.x].m[0] = 0;
        sdata[threadIdx.x].m[1] = 0;
        sdata[threadIdx.x].m[2] = 0;
        sdata[threadIdx.x].m[3] = 0;
    }
    __syncthreads();

    int if_carry[4] = {0,0,0,0};
    for(int stride = blockDim.x / 2;stride > 0;stride >>= 1){
        if (threadIdx.x < stride) {
            if_carry[0] = 0;
            if_carry[1] = 0;
            if_carry[2] = 0;
            if_carry[3] = 0;
            //m3
            if(sdata[threadIdx.x + stride].m[3] > (uint64_t)0xffffffffffffffff - sdata[threadIdx.x].m[3]) if_carry[3] = 1;
            sdata[threadIdx.x].m[3] += sdata[threadIdx.x + stride].m[3];
            //if m3 carry
            if(sdata[threadIdx.x + stride].m[2] > (uint64_t)0xffffffffffffffff - if_carry[3]) if_carry[2] = 1;
            sdata[threadIdx.x].m[2] += if_carry[3];
            //m2
            if(sdata[threadIdx.x + stride].m[2] > (uint64_t)0xffffffffffffffff - sdata[threadIdx.x].m[2]) if_carry[2] = 1;
            sdata[threadIdx.x].m[2] += sdata[threadIdx.x + stride].m[2];
            //if m2 carry
            if(sdata[threadIdx.x + stride].m[1] > (uint64_t)0xffffffffffffffff - if_carry[2]) if_carry[1] = 1;
            sdata[threadIdx.x].m[1] += if_carry[2];
            //m1
            if(sdata[threadIdx.x + stride].m[1] > (uint64_t)0xffffffffffffffff - sdata[threadIdx.x].m[1]) if_carry[1] = 1;
            sdata[threadIdx.x].m[1] += sdata[threadIdx.x + stride].m[1];
            //if m1 carry
            if(sdata[threadIdx.x + stride].m[0] > (uint64_t)0xffffffffffffffff - if_carry[1]) if_carry[0] = 1;
            sdata[threadIdx.x].m[0] += if_carry[1];
            if(sdata[threadIdx.x + stride].m[0] > (uint64_t)0xffffffffffffffff - sdata[threadIdx.x].m[0]) if_carry[0] = 1;
            sdata[threadIdx.x].m[0] += sdata[threadIdx.x + stride].m[0];
        }
        __syncthreads();
    }

    if(threadIdx.x==0){
        output[blockIdx.x].m[0] = sdata[0].m[0];
        output[blockIdx.x].m[1] = sdata[0].m[1];
        output[blockIdx.x].m[2] = sdata[0].m[2];
        output[blockIdx.x].m[3] = sdata[0].m[3];
    }
    return;
}